/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;
typedef double var_type;
#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005
#define MAX_X_DIM 32
#define MAX_Y_DIM 32

#define checkCudaErrors(ans) {                                \
          hipError_t err = ans;                                   \
          if (err != hipSuccess) {                                 \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
                  hipGetErrorString(err)); \
            hipDeviceReset();                                      \
            exit(EXIT_FAILURE);                                     \
          }                                                          \
        }                                                         \

//------------------------------DEVICE_CODE------------------------------

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////

__global__ void convolutionRowGPU(var_type *d_Dst, var_type *d_Src, var_type *d_Filter, 
                       int imageW, int imageH, int filterR) {

    int k;

      int x =  blockIdx.x*blockDim.x + threadIdx.x;   
      int y =  blockIdx.y*blockDim.y +threadIdx.y;

    var_type sum = 0;

    for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += d_Src[y * imageW + d] * d_Filter[filterR - k];
        }    
    } 
    d_Dst[y * imageW + x] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////

__global__ void convolutionColumnGPU(var_type *d_Dst, var_type *d_Src, var_type *d_Filter,
    			   int imageW, int imageH, int filterR) {

    int k;

    int x =  blockIdx.x*blockDim.x + threadIdx.x;   
    int y =  blockIdx.y*blockDim.y +threadIdx.y;
                      
    var_type sum = 0;

    for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += d_Src[d * imageW + x] * d_Filter[filterR - k];
        }   
    }  
    d_Dst[y * imageW + x] = sum;
}

//-----------------------------------------------------------------------

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(var_type *h_Dst, var_type *h_Src, var_type *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      var_type sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(var_type *h_Dst, var_type *h_Src, var_type *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      var_type sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    var_type
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU,
    *h_OutputGPU,
    acc = accuracy;

    float elapsed= 0.0f;
    hipEvent_t start, stop;

    int imageW;
    int imageH;
    unsigned int i;

    //dim3 threads(imageH,imageW);


	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

     //Each block can have up to 32*32 threads. Thus, each block can process up to 32*32 pixels
    dim3 threads(MAX_X_DIM,MAX_Y_DIM);

    //Each blocks processes 32*32 pixels, and there are (imageW / 32) * (imageH / 32) blocks
    dim3 blocks(imageH/MAX_X_DIM,imageW/MAX_Y_DIM);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (var_type *)malloc(FILTER_LENGTH * sizeof(var_type));
    h_Input     = (var_type *)malloc(imageW * imageH * sizeof(var_type));
    h_Buffer    = (var_type *)malloc(imageW * imageH * sizeof(var_type));
    h_OutputCPU = (var_type *)malloc(imageW * imageH * sizeof(var_type));
    h_OutputGPU = (var_type *)malloc(imageW * imageH * sizeof(var_type));

    //Check for malloc success
    if(h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputCPU == NULL || h_OutputGPU == NULL){
        fprintf(stderr, "Failed to allocate host variables!\n");
        exit(EXIT_FAILURE);
    }

    //allocate device (GPU) memory
    checkCudaErrors( hipMallocManaged((void**)&d_Filter,FILTER_LENGTH * sizeof(var_type)) );
    checkCudaErrors( hipMallocManaged((void**)&d_Input,imageH * imageW * sizeof(var_type)) );
    checkCudaErrors( hipMallocManaged((void**)&d_Buffer,imageH * imageW * sizeof(var_type)) );
    checkCudaErrors( hipMallocManaged((void**)&d_OutputGPU,imageH * imageW * sizeof(var_type)) );

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);
    // initialization of host arrays
    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (var_type)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (var_type)rand() / ((var_type)RAND_MAX / 255) + (var_type)rand() / (var_type)RAND_MAX;
    }

    //GPU Computation
    printf("GPU computation...\n");
    
    checkCudaErrors( hipEventCreate(&start) );
    checkCudaErrors( hipEventCreate(&stop) );

    checkCudaErrors( hipEventRecord(start, 0) );

    //initialization of device arrays
    checkCudaErrors( hipMemcpy(d_Filter, h_Filter,FILTER_LENGTH * sizeof(var_type),hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy(d_Input, h_Input,imageH * imageW * sizeof(var_type),hipMemcpyHostToDevice) );

    convolutionRowGPU<<<blocks,threads>>>(d_Buffer,d_Input,d_Filter,imageW,imageH,filter_radius);

    hipError_t error=hipGetLastError();

    checkCudaErrors(error);
    
    hipDeviceSynchronize();
    
    convolutionColumnGPU<<<blocks,threads>>>(d_OutputGPU,d_Buffer,d_Filter,imageW,imageH,filter_radius);

    error=hipGetLastError();
    checkCudaErrors(error);

    checkCudaErrors( hipMemcpy(h_OutputGPU,d_OutputGPU,imageH * imageW * sizeof(var_type),hipMemcpyDeviceToHost) );
    
    checkCudaErrors( hipEventRecord(stop, 0) );
    checkCudaErrors( hipEventSynchronize (stop) );
    checkCudaErrors( hipEventElapsedTime(&elapsed, start, stop) );

    printf("GPU computation: COMPLETED!\n");

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    clock_t cpu_startTime, cpu_endTime;
    
    cpu_startTime = clock();

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    printf("CPU computation: COMPLETED!\n");

    cpu_endTime = clock();

    var_type cpuElapsedTime =  (var_type(cpu_endTime - cpu_startTime) ) / CLOCKS_PER_SEC;

    printf("GPU Elapsed Time: %.3f msec\n", elapsed);
    printf("CPU Elapsed Time: %10.8f sec\n", cpuElapsedTime);
    
    printf("Checking computed result for correctness: \n");
    bool correct = true;

    for (i = 0; i < imageW * imageH; i++) {
	while(1){
        	if(h_OutputGPU[i] > h_OutputCPU[i] + acc || h_OutputGPU[i] < h_OutputCPU[i] - acc){
            		printf("Error: h_OutputGPU[%d] = %f, with difference: %f\n", i, h_OutputGPU[i], h_OutputGPU[i] - h_OutputCPU[i]);
            		correct = false;
         		acc += acc + 0.00001;
			continue; 
        	}
		else
			break;
	}
    }

    printf("accuracy: %f\n", acc);

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    //Clean up CPU Memory
    free(h_OutputCPU);
    free(h_OutputGPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    //Clean up GPU Memory
    checkCudaErrors( hipFree(d_OutputGPU) );
    checkCudaErrors( hipFree(d_Buffer) );
    checkCudaErrors( hipFree(d_Input) );
    checkCudaErrors( hipFree(d_Filter) );
    checkCudaErrors( hipEventDestroy(start) );
    checkCudaErrors( hipEventDestroy(stop) );

    hipDeviceReset();

    if(correct) 
        return 0;

    else
        return 1;


    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  



    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    // cudaDeviceReset();


    return 0;
}
