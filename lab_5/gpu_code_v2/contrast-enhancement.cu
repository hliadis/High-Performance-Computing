#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "hist-equ.h"
#define checkCudaErrors(ans) {                                \
          hipError_t err = ans;                                   \
          if (err != hipSuccess) {                                 \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
                  hipGetErrorString(err)); \
            hipDeviceReset();                                      \
            exit(EXIT_FAILURE);                                     \
          }                                                          \
        }                                                         \

PGM_IMG contrast_enhancement_g(PGM_IMG img_in)
{   //int hist[256];
    PGM_IMG result;
    result.w = img_in.w;
    result.h = img_in.h;
    
    //histogram(hist, img_in.img, img_in.h * img_in.w, 256);
    
    checkCudaErrors (hipHostAlloc (&result.img, result.w * result.h * sizeof(unsigned char),0));
    
    histogram_equalization(result.img,img_in.img,result.w*result.h, 256);
    return result;
}

