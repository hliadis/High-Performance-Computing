/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;
#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005
#define MAX_X_DIM 32
#define MAX_Y_DIM 32
__constant__ double d_Filter[257];
#define CPU

#define checkCudaErrors(ans) {                                \
          hipError_t err = ans;                                   \
          if (err != hipSuccess) {                                 \
            printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
                  hipGetErrorString(err)); \
            hipDeviceReset();                                      \
            exit(EXIT_FAILURE);                                     \
          }                                                          \
        }                                                         \

//------------------------------DEVICE_CODE------------------------------

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////

__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, int pad_imageW, int filterR) {

    int k;

      int x =  blockIdx.x*blockDim.x + threadIdx.x +filterR;   
      int y =  blockIdx.y*blockDim.y +threadIdx.y + filterR;
      int idx = y *(pad_imageW) + x;
      int x_Dim = 2*filterR + blockDim.x;

      extern __shared__ double s_Input[];

      for(int k = 0; k + threadIdx.x < x_Dim ; k+= blockDim.x){
	      s_Input[threadIdx.y*x_Dim + threadIdx.x + k] = d_Src[idx - filterR + k];
      }

      __syncthreads();

    double sum = 0;
    
    for (k = -filterR; k <= filterR; k++) {
        sum += s_Input[threadIdx.y*x_Dim + threadIdx.x + filterR + k] * d_Filter[filterR - k];
    }

    d_Dst[idx] = sum; 
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////

__global__ void convolutionColumnGPU(double *d_Dst, double *d_Src, int pad_imageH ,int filterR) {
    
    int k;

    int x =  blockIdx.x*blockDim.x + threadIdx.x + filterR;   
    int y =  blockIdx.y*blockDim.y +threadIdx.y + filterR;
    int idx = y*pad_imageH + x;
    int y_Dim = 2*filterR + blockDim.y;
    int dst_idx = (y-filterR)*(pad_imageH - 2*filterR) + (x-filterR);

    extern __shared__ double s_Input[];

    for(int k = 0; k + threadIdx.y < y_Dim; k+= blockDim.y ){
        s_Input[threadIdx.y * blockDim.x + k*blockDim.x + threadIdx.x] = d_Src[idx + (k - filterR)*pad_imageH];
    }

    __syncthreads();
    
    double sum = 0;

    for (k = -filterR; k <= filterR; k++) {
        sum += s_Input[(threadIdx.y * blockDim.x) + (filterR + k)*blockDim.x + threadIdx.x] * d_Filter[filterR - k];
    }  

    d_Dst[dst_idx] = sum;
}

//-----------------------------------------------------------------------

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH,  int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
        
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH,  int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

/*void case_free(void *d_Input, void *d_OutputGPU, void *h_Filter, void *h_Buffer, void *h_OutputCPU, void * h_OutputGPU, void *h_temp, void *h_temp2, cudaError_t error){

  if(error != cudaSuccess){
      //Clean up CPU Memory
    free(h_OutputCPU);
    free(h_OutputGPU);
    free(h_Buffer);
    free(h_temp);
    free(h_temp2);
    free(h_Input);
    free(h_Filter);

    //Clean up GPU Memory
    cudaFree(d_OutputGPU);
    cudaFree(d_Input);

  }


}*/

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,

#ifdef CPU
    *h_Buffer,
    *h_OutputCPU,
    *h_Input,
#endif

    *h_temp,
    *h_temp2,
    *d_Input0,
    *d_Input1,
    *d_OutputGPU0,
    *d_OutputGPU1,
    *h_OutputGPU;

    float elapsed= 0.0f;
    hipEvent_t start, stop;

    int imageW;
    int imageH;
    int pad_imageW;
    int pad_imageH;
    int block_size;
    int chunk;
    int tile_idx;
    int pos;

    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    dim3 grid, block;

    unsigned int i, padding_size;
    int posx, posy;

	printf("Enter filter radius: ");
	if (scanf("%d", &filter_radius) == EOF){
    fprintf(stderr,"Invalid Input!\n");
    exit(EXIT_FAILURE);
  }

    padding_size = 2*filter_radius;

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    if (scanf("%d", &imageW) == EOF){
      fprintf(stderr,"Invalid Input!\n");
      exit(EXIT_FAILURE);
    }

    imageH = imageW;

    printf(" Enter block size (should be a power of 2): ");
    if(scanf("%d", &block_size) == EOF){
      fprintf(stderr,"Invalid Input!\n");
      exit(EXIT_FAILURE);
    }

    chunk = imageH*imageW / block_size;

    //Determine padded array size
    pad_imageW = padding_size + imageW;
    pad_imageH = padding_size + imageH;


    if(imageW > MAX_X_DIM){
      block.x = MAX_X_DIM;
      grid.x = imageW/block.x;
      //
      if(block_size/imageW > MAX_X_DIM){
        block.y = MAX_Y_DIM;
        grid.y = (block_size/imageW)/MAX_Y_DIM;
      }

      else{
        block.y = block_size/imageW;
        grid.y = 1;
      }

      tile_idx = block_size/imageW;
    }

    else{
      grid =1;
      block.x = imageW;
      block.y = imageH;
      chunk = 1;
      tile_idx = imageH;
    }


    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    //h_temp = (double *)malloc(pad_imageW * pad_imageH * sizeof(double));
    //h_temp2 = (double *)malloc(pad_imageW * pad_imageH * sizeof(double));
    
#ifdef CPU
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
#endif
    /*checkCudaErrors(cudaHostAlloc((void **) &h_OutputGPU, imageH * imageW * sizeof(double), cudaHostAllocMapped));
    checkCudaErrors(cudaHostAlloc((void **) &h_temp, pad_imageH * pad_imageW * sizeof(double), cudaHostAllocMapped));
    h_temp2 = h_temp;

    //Check for malloc success
    if(h_Filter == NULL){
        fprintf(stderr, "Failed to allocate host variable h_Filter!\n");
        exit(EXIT_FAILURE);
    }
    else if(h_temp == NULL){
          fprintf(stderr, "Failed to allocate host variable h_temp!\n");
          exit(EXIT_FAILURE);
    }
    else if(h_OutputGPU == NULL){
      fprintf(stderr, "Failed to allocate host variable h_OutputGPU!\n");
          exit(EXIT_FAILURE);
    }*/

#ifdef CPU
    if(h_Buffer == NULL || h_OutputCPU == NULL  || h_Input == NULL){
        fprintf(stderr, "Failed to allocate host variables!\n");
        exit(EXIT_FAILURE);
    }
#endif

    //allocate device (GPU) memory
    checkCudaErrors( hipMalloc((void**)&d_Filter,FILTER_LENGTH * sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&d_Input0,pad_imageW * (filter_radius * 2 + tile_idx) * sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&d_Input1,pad_imageW * (filter_radius * 2 + tile_idx) * sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&d_OutputGPU0,pad_imageW * (filter_radius * 2 + tile_idx) * sizeof(double)) );
    checkCudaErrors( hipMalloc((void**)&d_OutputGPU1,pad_imageW * (filter_radius * 2 + tile_idx) * sizeof(double)) );
    checkCudaErrors(hipMemset(d_Input0, 0, pad_imageW*(filter_radius * 2 + tile_idx) *sizeof(double)));
    checkCudaErrors(hipMemset(d_OutputGPU0, 0, pad_imageW*(filter_radius * 2 + tile_idx) *sizeof(double)));
    checkCudaErrors(hipMemset(d_Input1, 0, pad_imageW*(filter_radius * 2 + tile_idx) *sizeof(double)));
    checkCudaErrors(hipMemset(d_OutputGPU1, 0, pad_imageW*(filter_radius * 2 + tile_idx) *sizeof(double)));

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);
    // initialization of host arrays
    checkCudaErrors(hipHostAlloc((void **) &h_OutputGPU, imageH * imageW * sizeof(double), hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc((void **) &h_temp, pad_imageH * pad_imageW * sizeof(double), hipHostMallocMapped));
    h_temp2 = h_temp;
    //memset((void*)h_temp2,0,(size_t)((pad_imageW*pad_imageH)*(size_t)sizeof(double)));

    //Check for malloc success
    if(h_Filter == NULL){
        fprintf(stderr, "Failed to allocate host variable h_Filter!\n");
        exit(EXIT_FAILURE);
    }
    else if(h_temp == NULL){
          fprintf(stderr, "Failed to allocate host variable h_temp!\n");
          exit(EXIT_FAILURE);
    }
    else if(h_OutputGPU == NULL){
      fprintf(stderr, "Failed to allocate host variable h_OutputGPU!\n");
          exit(EXIT_FAILURE);
    }
    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    i = 0;

    for(posy=0; posy < pad_imageH; posy++){
        for(posx=0; posx < pad_imageW; posx++){

            if(posy < filter_radius || posy >= imageH + filter_radius){
                h_temp[posy*pad_imageW + posx] = 0;
                //since y coordinate is out of [filter_radius, imageH + filter_radius]
                //we do not have to check x coordinate
                continue;
            }

            if(posx < filter_radius || posx >= imageW + filter_radius){
                h_temp[posy*pad_imageW + posx] = 0;
            }

            else{

                h_temp[posy*pad_imageW + posx] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
            #ifdef CPU
                h_Input[i] = h_temp[posy*pad_imageW + posx];
            #endif
                i++;
            }
        }
    }


    //GPU Computation
    printf("GPU computation...\n");
    
    checkCudaErrors( hipEventCreate(&start) );
    checkCudaErrors( hipEventCreate(&stop) );
    checkCudaErrors( hipEventRecord(start, 0) );
    
    checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(d_Filter), h_Filter,FILTER_LENGTH * sizeof(double),0,hipMemcpyHostToDevice) );
    
    pos = 0;

    for(i=0; (i < (chunk/2) ) || (chunk == 1); i++){
      checkCudaErrors(hipMemcpyAsync(d_Input0, h_temp + pos, (pad_imageW)*(tile_idx+2*filter_radius)*sizeof(double),
                      hipMemcpyHostToDevice, stream0));
      if(chunk != 1)
        checkCudaErrors(hipMemcpyAsync(d_Input1, h_temp + pos + (pad_imageW * tile_idx) , (pad_imageW)*(tile_idx+2*filter_radius)*sizeof(double),
                        hipMemcpyHostToDevice, stream1));

      convolutionRowGPU<<<grid,block,(block.x + 2*filter_radius)*block.y*sizeof(double),stream0>>>(d_OutputGPU0,d_Input0,pad_imageW, filter_radius);
      
      if(chunk != 1)
        convolutionRowGPU<<<grid,block,(block.x + 2*filter_radius)*block.y*sizeof(double),stream1>>>(d_OutputGPU1,d_Input1,pad_imageW, filter_radius);

      checkCudaErrors(hipMemcpyAsync(h_temp2 + pad_imageW*filter_radius + pos, d_OutputGPU0 + pad_imageW*filter_radius, pad_imageW * tile_idx * sizeof(double),
                      hipMemcpyDeviceToHost,stream0));
      if(chunk == 1)
        break;

      checkCudaErrors(hipMemcpyAsync(h_temp2 + pad_imageW*filter_radius + pos + (pad_imageW * tile_idx) , d_OutputGPU1 + pad_imageW*filter_radius, pad_imageW * tile_idx * sizeof(double),
                      hipMemcpyDeviceToHost,stream1));

      pos += 2*(pad_imageW * tile_idx);
    }

    hipDeviceSynchronize();

    pos = 0;
    int final_pos = 0;

    for(i=0; (i < (chunk/2) ) || (chunk == 1); i++){
      checkCudaErrors(hipMemcpyAsync(d_Input0, h_temp2 + pos, (pad_imageW)*(tile_idx+2*filter_radius)*sizeof(double), 
                      hipMemcpyHostToDevice, stream0));
      
      if(chunk != 1)
        checkCudaErrors(hipMemcpyAsync(d_Input1, h_temp2 + pos + (pad_imageW * tile_idx) , (pad_imageW)*(tile_idx+2*filter_radius)*sizeof(double), 
                        hipMemcpyHostToDevice, stream1));

      convolutionColumnGPU<<<grid,block,(block.y + 2*filter_radius)*block.x*sizeof(double),stream0>>>(d_OutputGPU0,d_Input0,pad_imageH,filter_radius);

      if(chunk != 1)
        convolutionColumnGPU<<<grid,block,(block.y + 2*filter_radius)*block.x*sizeof(double),stream1>>>(d_OutputGPU1,d_Input1,pad_imageH,filter_radius);

      
      checkCudaErrors(hipMemcpyAsync(h_OutputGPU + final_pos, d_OutputGPU0, imageW * tile_idx *sizeof(double), 
                      hipMemcpyDeviceToHost,stream0));
      if(chunk == 1)
        break;

      checkCudaErrors(hipMemcpyAsync(h_OutputGPU + final_pos + (imageW * tile_idx) , d_OutputGPU1, imageW * tile_idx *sizeof(double), 
                      hipMemcpyDeviceToHost,stream1));

      pos += 2*(pad_imageW * tile_idx);
      final_pos += 2*(imageW * tile_idx);
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);

    checkCudaErrors( hipEventRecord(stop, 0) );
    checkCudaErrors( hipEventSynchronize (stop) );
    checkCudaErrors( hipEventElapsedTime(&elapsed, start, stop) );

    printf("GPU computation: COMPLETED!\n");

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.

#ifdef CPU
    printf("CPU computation...\n");

    clock_t cpu_startTime, cpu_endTime;
    
    cpu_startTime = clock();

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
    printf("CPU computation: COMPLETED!\n");

    cpu_endTime = clock();

    double cpuElapsedTime =  (double(cpu_endTime - cpu_startTime) ) / CLOCKS_PER_SEC;
#endif

    printf("GPU Elapsed Time: %.3f msec\n", elapsed);

    bool correct = true;

#ifdef CPU
    printf("CPU Elapsed Time: %10.8f sec\n", cpuElapsedTime);

    printf("Checking computed result for correctness: \n");

    for (i = 0; i < imageW * imageH; i++) {
        if(h_OutputGPU[i] > h_OutputCPU[i] + accuracy || h_OutputGPU[i] < h_OutputCPU[i] - accuracy){
            printf("Error: h_OutputGPU[%d] = %f, with difference: %f\n", i, h_OutputGPU[i], h_OutputGPU[i] - h_OutputCPU[i]);
            correct = false;
            break;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    //Clean up CPU Memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
#endif
    //free(h_temp);
    //free(h_temp2);
    checkCudaErrors( hipHostFree(h_OutputGPU) );
    checkCudaErrors( hipHostFree(h_temp) );
    free(h_Filter);

    //Clean up GPU Memory
    checkCudaErrors( hipFree(d_OutputGPU0) );
    checkCudaErrors( hipFree(d_OutputGPU1) );
    //checkCudaErrors( cudaFree(d_Buffer) );
    checkCudaErrors( hipFree(d_Input0) );
    checkCudaErrors( hipFree(d_Input1) );
    //checkCudaErrors( cudaFree(d_Filter) );
    checkCudaErrors( hipEventDestroy(start) );
    checkCudaErrors( hipEventDestroy(stop) );

    hipDeviceReset();

    if(correct) 
        return 0;

    else
        return 1;
}
